#include "hip/hip_runtime.h"
// #include <stdlib.h>
#include <stdio.h>
#include "utilities.h"
// #include <cmath>
// const float infinity = INFINITY;

//ToDo
//kernel execution
//copy distance values and check
//remove cls_h and associated read functions


__global__ void calc_dist_texture(hipTextureObject_t queryP,
                                  int n_queryP,
                                  float *refP,
                                  int n_refP,
                                  int ref_pitch,
                                  int n_dim,
                                  float *dist,
                                  int dist_pitch)
{
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < n_refP && yIndex < n_queryP)
    {
        float ssd = 0.f;
        for (int i = 0; i < n_dim; i++)
        {
            // float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            float tmp = refP[i * ref_pitch + xIndex] - tex2D<float>(queryP, (float)yIndex, (float)i);
            ssd += tmp * tmp;
        }
        // dist[yIndex * query_pitch + xIndex] = ssd;
        // dist[yIndex * ref_pitch + xIndex] = ssd;
        dist[yIndex * dist_pitch + xIndex] = ssd;

    }else if(yIndex < n_queryP)
    {
        // dist[yIndex * ref_pitch + xIndex] = infinity;
        dist[yIndex * dist_pitch + xIndex] = infinity;
    }
    
}




// __global__ void calc_dist_texture(hipTextureObject_t queryP,
//                                   int n_queryP,
//                                   float *refP,
//                                   int n_refP,
//                                   int ref_pitch,
//                                   int n_dim,
//                                   float *dist)
// {
//     unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
//     unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
//     unsigned int tIdx = threadIdx.x;
//     unsigned int blockDimx = blockDim.x;

//     __shared__ float sharedRef[64];

//     if(yIndex % blockDim.y == 0){

//         for (size_t i = 0; i < n_dim; i++)
//         {
//             sharedRef[i * blockDimx + tIdx] = refP[i * ref_pitch + xIndex];
//         } 
//     }
//     __syncthreads();

//     if (xIndex < n_refP && yIndex < n_queryP)
//     {
//         float ssd = 0.f;
//         for (int i = 0; i < n_dim; i++)
//         {
//             // float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
//             // float tmp = refP[i * ref_pitch + xIndex] - tex2D<float>(queryP, (float)yIndex, (float)i);
//             float tmp = sharedRef[i * blockDimx + tIdx] - tex2D<float>(queryP, (float)yIndex, (float)i);
//             ssd += tmp * tmp;
//         }
//         // dist[yIndex * query_pitch + xIndex] = ssd;
//         dist[yIndex * ref_pitch + xIndex] = ssd;
//     }
// }



__global__ void sort_dist_bitonic(float *distances, int *indexes, int n_refP, int dist_pitch, int n_queryP, const uint stage, const uint passOfStage)
{

    // uint threadId = get_global_id(0);
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < n_refP / 2 && yIndex < n_queryP)
    {

        unsigned int pairDistance = 1 << (stage - passOfStage);
        unsigned int blockWidth = 2 * pairDistance;
        unsigned int temp;
        bool compareResult;

        unsigned int leftId = (xIndex & (pairDistance - 1)) + (xIndex >> (stage - passOfStage)) * blockWidth;
        unsigned int rightId = leftId + pairDistance;

        float leftElement, rightElement;
        float greater, lesser;
        int left_idx, right_idx, greater_idx, lesser_idx;

        // leftElement = distances[yIndex * n_refP + leftId];
        // rightElement =distances[yIndex * n_refP +rightId];

        // leftElement_cls = clases[yIndex * n_refP + leftId];
        // rightElement_cls = clases[yIndex * n_refP +rightId];

        leftElement = distances[yIndex * dist_pitch + leftId];
        rightElement = distances[yIndex * dist_pitch + rightId];

        if (stage == 0 && passOfStage == 0)
        {
            left_idx = leftId;
            right_idx = rightId;
        }
        else
        {

            left_idx = indexes[yIndex * dist_pitch + leftId];
            right_idx = indexes[yIndex * dist_pitch + rightId];
        }

        // leftElement_cls = clases[yIndex * dist_pitch + leftId];
        // rightElement_cls = clases[yIndex * dist_pitch +rightId];

        unsigned int sameDirectionBlockWidth = xIndex >> stage;
        unsigned int sameDirection = sameDirectionBlockWidth & 0x1;

        temp = sameDirection ? rightId : temp;
        rightId = sameDirection ? leftId : rightId;
        leftId = sameDirection ? temp : leftId;

        compareResult = (leftElement < rightElement);

        /////////////////////////////////////////////////////////////////////////////
        /*add these to a single if else block*/
        greater = compareResult ? rightElement : leftElement;
        // greater_cls = compareResult ? rightElement_cls : leftElement_cls;
        greater_idx = compareResult ? right_idx : left_idx;

        lesser = compareResult ? leftElement : rightElement;
        // lesser_cls = compareResult ? leftElement_cls : rightElement_cls;
        lesser_idx = compareResult ? left_idx : right_idx;
        //////////////////////////////////////////////////////////////////////////////

        distances[yIndex * dist_pitch + leftId] = lesser;
        distances[yIndex * dist_pitch + rightId] = greater;

        //dist_pitch = cls_pitch
        // clases[yIndex * dist_pitch + leftId] = lesser_cls;
        // clases[yIndex * dist_pitch +rightId] = greater_cls;
        // if (xIndex == 0 && yIndex == 0)
        //     printf("lesser idx : %d", left_idx);

        indexes[yIndex * dist_pitch + leftId] = lesser_idx;
        indexes[yIndex * dist_pitch + rightId] = greater_idx;
    }
}


// unsigned int  getNearestIntOfPow2(int n){

//     if(!(n&(n-1))){//if n is already a power of 2
//         return n;
//     }else
//     {
//         int bitIndex = 0;//equal to log2
//         int shift = 0;
//         // int a[5] = {}

//         bitIndex = (n>0xFFFF) << 4;
//         n >>= bitIndex;

//         shift = (n>0xFF) << 3;
//         n >>= shift;
//         bitIndex |= shift;

//         shift = (n>0xF) << 2;
//         n >>= shift; 
//         bitIndex |= shift;

//         shift = (n>0x3) << 1;
//         n >>= shift; 
//         bitIndex |= shift;

//         bitIndex |= (n >> 1);

//         return 1 << (bitIndex+1);

//     }
    
// }

bool knn_cuda_texture_new(const float *ref_h,
                      int n_refPoints_original,
                      const float *query_h,
                      int n_queryPoints,
                      int n_dimentions,
                      int k,
                      float *dist_h,
                      int *idx_h)
{

    hipError_t error;
    hipDeviceProp_t prop;
    int n_devices;
    int warpSize = 32;

    unsigned int n_refPoints = getNearestIntOfPow2(n_refPoints_original);

    // printf("\n(texture new)after getNearestPower....\n");

    error = hipGetDeviceCount(&n_devices);
    if (error != hipSuccess || n_devices == 0)
    {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    error = hipSetDevice(0);
    if (error != hipSuccess)
    {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    error = hipGetDeviceProperties(&prop, 0);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        // exit(-1);
        return false;
    }

    warpSize = prop.warpSize;

    // Allocate global memory
    float *ref_dev = NULL;
    float *dist_dev = NULL;
    int *idx_dev = NULL;

    size_t ref_pitch_in_bytes;
    size_t dist_pitch_in_bytes;
    size_t idx_pitch_in_bytes;

    // printf("\n(texture new)before hipMallocPitch");

    error = hipMallocPitch((void **)&ref_dev, &ref_pitch_in_bytes, n_refPoints_original * sizeof(float), n_dimentions);
    error = hipMallocPitch((void **)&dist_dev, &dist_pitch_in_bytes, n_refPoints * sizeof(float), n_queryPoints);
    error = hipMallocPitch((void **)&idx_dev, &idx_pitch_in_bytes, n_refPoints * sizeof(int), n_queryPoints);

    // printf("\n(texture new)after hipMallocPitch\n");

    if (error != hipSuccess)
    {
        printf("Error in hipMallocPitch: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);

        return false;
    }
    ///////check whether all pitch are equal

    // Deduce pitch value of reference points
    size_t ref_pitch = ref_pitch_in_bytes / sizeof(float);
    size_t dist_pitch = dist_pitch_in_bytes / sizeof(float);
    size_t idx_pitch = idx_pitch_in_bytes / sizeof(int);

    // printf("\n(texture new)ref_pitch: %d\n", ref_pitch);
    // printf("\n(texture new)dist_pitch: %d\n", dist_pitch);
    // printf("\n(texture new)idx_pitch: %d\n", idx_pitch);
    

    // printf("\n(texture new)before hipMemcpy2D.....\n");

    //copy ref data from host to device (in column major)
    // error = hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_h, n_refPoints * sizeof(float), n_refPoints * sizeof(float), n_dimentions, hipMemcpyHostToDevice);
    error = hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_h, n_refPoints_original * sizeof(float), n_refPoints_original * sizeof(float), n_dimentions, hipMemcpyHostToDevice);

    // printf("\n(texture new)after hipMemcpy2D.....\n");

    if (error != hipSuccess)
    {

        printf("Error in hipMemcpy2D: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);

        return false;
    }

    // Allocate CUDA array for query points
    hipArray *query_array_dev = NULL;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    error = hipMallocArray(&query_array_dev, &channel_desc, n_queryPoints, n_dimentions);

    if (error != hipSuccess)
    {

        printf("Error in hipMallocArray: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);

        return false;
    }

    // Copy query points from host to device
    error = hipMemcpyToArray(query_array_dev, 0, 0, query_h, n_queryPoints * sizeof(float) * n_dimentions, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {

        printf("Error in hipMemcpyToArray: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);

        return false;
    }

    // Resource descriptor
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = query_array_dev;

    // Texture descriptor
    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0] = hipAddressModeClamp;
    tex_desc.addressMode[1] = hipAddressModeClamp;
    tex_desc.filterMode = hipFilterModePoint;
    tex_desc.readMode = hipReadModeElementType;
    tex_desc.normalizedCoords = 0;

    hipTextureObject_t query_tex_dev = 0;
    error = hipCreateTextureObject(&query_tex_dev, &res_desc, &tex_desc, NULL);

    if (error != hipSuccess)
    {

        printf("Error in hipCreateTextureObject: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);

        return false;
    }

    // printf("\n(texture new)before grid block sizes...\n");

    /////only considered >16
    int block_size_x = warpSize / 2;
    int block_size_y = warpSize / 2;
    int grid_size_x = n_refPoints / block_size_x;
    int grid_size_y = n_queryPoints / block_size_y;

    dim3 block_size = dim3(block_size_x, block_size_y);
    dim3 grid_size = dim3(grid_size_x, grid_size_y);

    // printf("\n(texture new)start dist calc\n");

    calc_dist_texture<<<grid_size, block_size>>>(query_tex_dev, n_queryPoints, ref_dev, n_refPoints_original, ref_pitch, n_dimentions, dist_dev, dist_pitch);

    // hipDeviceSynchronize();
    hipDeviceSynchronize();

    // printf("\n(texture new)finished dist calc\n");

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        // printf("error in calc_dist_texture\n");
        printf("Error in calc_dist_texture kernel: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);
        hipDestroyTextureObject(query_tex_dev);

        return false;
    }

    //////////////////////////////////////////////////////////////////////////////////

    // printf("\n(texture new)before hipMemcpy2D \n");

    //remove after test.....
    
    // error = hipMemcpy2D(dist_h, n_refPoints * sizeof(float), dist_dev, dist_pitch_in_bytes, n_refPoints * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);
    // error = hipMemcpy2D(dist_h, n_refPoints_original * sizeof(float), dist_dev, dist_pitch_in_bytes, n_refPoints_original * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);

    // printf("\n(texture new)after hipMemcpy2D \n");

    if (error != hipSuccess)
    {
        // printf("error in calc_dist_texture\n");
        printf("Error hipMemcpy2D hipMemcpyDeviceToHost after calc_dist_texture kernel execution: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);
        hipDestroyTextureObject(query_tex_dev);

        return false;
    }


    //  printf("\n\ndistances before sort\n");
    // for(int i = 0; i<n_refPoints ; i++){

    //     printf("%d)%f  ", i,dist_h[0 + i]);

    // }

    ////////////////////////////////////////////////////////////////////////////////////////

    block_size_x = (n_refPoints / 2) / warpSize;
    block_size_y = n_queryPoints / warpSize;

    // block_size = dim3(warpSize, warpSize);
    block_size = dim3(warpSize, warpSize);
    grid_size = dim3(block_size_x, block_size_y);



    //////////////////////////////////////////////////

     unsigned int numStages = 0, stage = 0, passOfStage = 0, temp = 0;

    for (temp = n_refPoints; temp > 1; temp >>= 1)
    {
        ++numStages;
    }

    for (stage = 0; stage < numStages; ++stage)
    {

        for (passOfStage = 0; passOfStage < stage + 1; ++passOfStage)
        {

            // sort_dist_bitonic<<<grid_size, block_size>>>(distances_d, clases_d, n_refPoints, n_queryPoints, stage, passOfStage);
            sort_dist_bitonic<<<grid_size, block_size>>>(dist_dev, idx_dev, n_refPoints, dist_pitch, n_queryPoints, stage, passOfStage);
            hipDeviceSynchronize();
        }
    }

    error = hipGetLastError();

    if (error != hipSuccess)

    {
        // printf("error in sort kernel\n");
        printf("Error in sort_dist_bitonic kernel: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);
        hipDestroyTextureObject(query_tex_dev);

        return false;
    }

    // error = hipMemcpy2D(dist_h, n_refPoints * sizeof(float), dist_dev, dist_pitch_in_bytes, n_refPoints * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);
    error = hipMemcpy2D(dist_h, k * sizeof(float), dist_dev, dist_pitch_in_bytes, k * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);
    // error = hipMemcpy2D(idx_h,  n_refPoints * sizeof(int), idx_dev,  idx_pitch_in_bytes,  n_refPoints * sizeof(int), n_queryPoints, hipMemcpyDeviceToHost);
    error = hipMemcpy2D(idx_h, k * sizeof(int), idx_dev, idx_pitch_in_bytes, k * sizeof(int), n_queryPoints, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        // printf("error in hipMemcpy2D\n");
        printf("Error in hipMemcpy2D hipMemcpyDeviceToHost after sort_dist_bitonic kernel execution: %s\n", hipGetErrorString(error));
        // exit(-1);
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);
        hipDestroyTextureObject(query_tex_dev);

        return false;
    }

     // Memory clean-up
     hipFree(ref_dev);
     hipFree(dist_dev);
     hipFree(idx_dev);
     hipFreeArray(query_array_dev);
     hipDestroyTextureObject(query_tex_dev);
 
     return true;

}




// int main()
// {

//     int n_refPoints = 8192;
//     int n_queryPoints = 1024;
//     int n_dimentions = 4;
//     int k = 4;

//     // char *refPointsFileName = "testData32_4.csv";
//     // char *queryPointsFileName = "queryPoints1_4.csv";
//     char *refPointsFileName = "testData8192_4.csv";
//     char *queryPointsFileName = "queryPoints_4.csv";
    

//     hipError_t error;
//     hipDeviceProp_t prop;
//     int device_count;
//     // int warpSize = 32;

//     error = hipGetDeviceCount(&device_count);

//     if (error != hipSuccess)
//     {
//         printf("Error: %s\n", hipGetErrorString(error));
//         exit(-1);
//     }

//     error = hipGetDeviceProperties(&prop, 0);

//     if (error != hipSuccess)
//     {
//         printf("Error: %s\n", hipGetErrorString(error));
//         exit(-1);
//     }

//     // warpSize = prop.warpSize;

    
//     float *ref_h = (float *)malloc(sizeof(float) * n_dimentions * n_refPoints);
//     float *query_h = (float *)malloc(sizeof(float) * n_dimentions * n_queryPoints);
//     float *dist_h = (float *)malloc(sizeof(float) * n_refPoints * n_queryPoints);
//     // float *dist_h = (float *)malloc(sizeof(float) * k * n_queryPoints);
//     int *idx_h = (int *)malloc(sizeof(int) * k * n_queryPoints);
//     // int *idx_h = (int *)malloc(sizeof(int) * n_refPoints * n_queryPoints);


//     float *ref_row_maj_h = (float *)malloc(sizeof(float) * n_dimentions * n_refPoints);
//     float *query_row_maj_h = (float *)malloc(sizeof(float) * n_dimentions * n_queryPoints);
    
  

//     readRefPoints(refPointsFileName, ref_row_maj_h, n_refPoints, n_queryPoints, n_dimentions);

//     ref_h = transpose(ref_row_maj_h, n_refPoints, n_dimentions); //make column major
//     free(ref_row_maj_h);


//     // for (int i = 0; i < noOfRefPoints; i++)
//     // for (int i = 0; i < 5; i++)
//     // {
//     //     printf("%d  %f  %f  %f  %f \n", i, ref_h[i * n_dimentions + 0], ref_h[i * n_dimentions + 1], ref_h[i * n_dimentions + 2], ref_h[i * n_dimentions + 3]);
//     // }

//     readQueryPoints(queryPointsFileName, query_row_maj_h, n_dimentions);
//     query_h = transpose(query_row_maj_h, n_queryPoints, n_dimentions); //make column major
//     free(query_row_maj_h);


//     // initialize_data(ref_h, n_refPoints, query_h, n_queryPoints, n_dimentions);


//     knn_cuda_texture_new(ref_h, n_refPoints, query_h, n_queryPoints, n_dimentions, k, dist_h, idx_h);

//     printf("\n\ndistances after sort....\n");
//     // for(int i=0; i< n_refPoints;i++){
//     for (int i = 0; i < k; i++)
//     {
//         // printf("%d)%d  ", i,idx_h[n_refPoints * 0 +i]);
//         printf("%d)%f  ", i, dist_h[k * 1000 + i]);
//     }

//     printf("\n\nindexes after sort....\n");
//     // for(int i=0; i< n_refPoints;i++){
//     for (int i = 0; i < k; i++)
//     {
//         // printf("%d)%d  ", i,idx_h[n_refPoints * 0 +i]);
//         printf("%d)%d  ", i, idx_h[k * 1000 + i]);
//     }

    
//     free(ref_h);
//     free(dist_h);
//     free(idx_h);
//     free(ref_row_maj_h);
//     free(query_row_maj_h);

//     return 0;
// }