#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utilities.h"
// #include <cmath>
// const float infinity = INFINITY;


__global__ void calc_dist_texture(hipTextureObject_t queryP,
                                  int n_queryP,
                                  float *refP,
                                  int n_refP,
                                  int ref_pitch,
                                  int n_dim,
                                  float *dist,
                                  int dist_pitch)
{
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < n_refP && yIndex < n_queryP)
    {
        float ssd = 0.f;
        for (int i = 0; i < n_dim; i++)
        {
            float tmp = refP[i * ref_pitch + xIndex] - tex2D<float>(queryP, (float)yIndex, (float)i);
            ssd += tmp * tmp;
        }
        
        dist[yIndex * dist_pitch + xIndex] = ssd;

    }else if(yIndex < n_queryP)
    {
        dist[yIndex * dist_pitch + xIndex] = infinity;
    }
    
}




// __global__ void calc_dist_texture(hipTextureObject_t queryP,
//                                   int n_queryP,
//                                   float *refP,
//                                   int n_refP,
//                                   int ref_pitch,
//                                   int n_dim,
//                                   float *dist)
// {
//     unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
//     unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
//     unsigned int tIdx = threadIdx.x;
//     unsigned int blockDimx = blockDim.x;

//     __shared__ float sharedRef[64];

////    if(threadIdx.y == 0)
//     if(yIndex % blockDim.y == 0){

//         for (size_t i = 0; i < n_dim; i++)
//         {
//             sharedRef[i * blockDimx + tIdx] = refP[i * ref_pitch + xIndex];
//         } 
//     }
//     __syncthreads();

//     if (xIndex < n_refP && yIndex < n_queryP)
//     {
//         float ssd = 0.f;
//         for (int i = 0; i < n_dim; i++)
//         {
//             // float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
//             // float tmp = refP[i * ref_pitch + xIndex] - tex2D<float>(queryP, (float)yIndex, (float)i);
//             float tmp = sharedRef[i * blockDimx + tIdx] - tex2D<float>(queryP, (float)yIndex, (float)i);
//             ssd += tmp * tmp;
//         }
//         // dist[yIndex * query_pitch + xIndex] = ssd;
//         dist[yIndex * ref_pitch + xIndex] = ssd;
//     }
// }



__global__ void sort_dist_bitonic(float *distances, int *indexes, int n_refP, int dist_pitch, int n_queryP, const uint stage, const uint passOfStage)
{

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < n_refP / 2 && yIndex < n_queryP)
    {

        unsigned int pairDistance = 1 << (stage - passOfStage);
        unsigned int blockWidth = 2 * pairDistance;
        unsigned int temp;
        bool compareResult;

        unsigned int leftId = (xIndex & (pairDistance - 1)) + (xIndex >> (stage - passOfStage)) * blockWidth;
        unsigned int rightId = leftId + pairDistance;

        float leftElement, rightElement;
        float greater, lesser;
        int left_idx, right_idx, greater_idx, lesser_idx;


        leftElement = distances[yIndex * dist_pitch + leftId];
        rightElement = distances[yIndex * dist_pitch + rightId];

        if (stage == 0 && passOfStage == 0)
        {
            left_idx = leftId;
            right_idx = rightId;
        }
        else
        {

            left_idx = indexes[yIndex * dist_pitch + leftId];
            right_idx = indexes[yIndex * dist_pitch + rightId];
        }

        
        unsigned int sameDirectionBlockWidth = xIndex >> stage;
        unsigned int sameDirection = sameDirectionBlockWidth & 0x1;

        temp = sameDirection ? rightId : temp;
        rightId = sameDirection ? leftId : rightId;
        leftId = sameDirection ? temp : leftId;

        compareResult = (leftElement < rightElement);
       
        greater = compareResult ? rightElement : leftElement;
        greater_idx = compareResult ? right_idx : left_idx;

        lesser = compareResult ? leftElement : rightElement;
        lesser_idx = compareResult ? left_idx : right_idx;

        distances[yIndex * dist_pitch + leftId] = lesser;
        distances[yIndex * dist_pitch + rightId] = greater;

        indexes[yIndex * dist_pitch + leftId] = lesser_idx;
        indexes[yIndex * dist_pitch + rightId] = greater_idx;
    }
}


//this method is included in utils.h

// unsigned int  getNearestIntOfPow2(int n){

//     if(!(n&(n-1))){//if n is already a power of 2
//         return n;
//     }else
//     {
//         int bitIndex = 0;//equal to log2
//         int shift = 0;
//         // int a[5] = {}

//         bitIndex = (n>0xFFFF) << 4;
//         n >>= bitIndex;

//         shift = (n>0xFF) << 3;
//         n >>= shift;
//         bitIndex |= shift;

//         shift = (n>0xF) << 2;
//         n >>= shift; 
//         bitIndex |= shift;

//         shift = (n>0x3) << 1;
//         n >>= shift; 
//         bitIndex |= shift;

//         bitIndex |= (n >> 1);

//         return 1 << (bitIndex+1);

//     }
    
// }

bool knn_cuda_texture_new(const float *ref_h,
                      int n_refPoints_original,
                      const float *query_h,
                      int n_queryPoints,
                      int n_dimentions,
                      int k,
                      float *dist_h,
                      int *idx_h)
{

    hipError_t error;
    hipDeviceProp_t prop;
    int n_devices;
    int warpSize = 32;

    unsigned int n_refPoints = getNearestIntOfPow2(n_refPoints_original);

    error = hipGetDeviceCount(&n_devices);
    if (error != hipSuccess || n_devices == 0)
    {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    error = hipSetDevice(0);
    if (error != hipSuccess)
    {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    error = hipGetDeviceProperties(&prop, 0);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        return false;
    }

    warpSize = prop.warpSize;

    // Allocate global memory
    float *ref_dev = NULL;
    float *dist_dev = NULL;
    int *idx_dev = NULL;

    size_t ref_pitch_in_bytes;
    size_t dist_pitch_in_bytes;
    size_t idx_pitch_in_bytes;

    error = hipMallocPitch((void **)&ref_dev, &ref_pitch_in_bytes, n_refPoints_original * sizeof(float), n_dimentions);
    error = hipMallocPitch((void **)&dist_dev, &dist_pitch_in_bytes, n_refPoints * sizeof(float), n_queryPoints);
    error = hipMallocPitch((void **)&idx_dev, &idx_pitch_in_bytes, n_refPoints * sizeof(int), n_queryPoints);


    if (error != hipSuccess)
    {
        printf("Error in hipMallocPitch: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);

        return false;
    }
    ///////check whether all pitch are equal

    // Deduce pitch value of reference points
    size_t ref_pitch = ref_pitch_in_bytes / sizeof(float);
    size_t dist_pitch = dist_pitch_in_bytes / sizeof(float);
    size_t idx_pitch = idx_pitch_in_bytes / sizeof(int);

    //copy ref data from host to device (in column major)
    error = hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_h, n_refPoints_original * sizeof(float), n_refPoints_original * sizeof(float), n_dimentions, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {

        printf("Error in hipMemcpy2D: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);

        return false;
    }

    // Allocate CUDA array for query points
    hipArray *query_array_dev = NULL;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    error = hipMallocArray(&query_array_dev, &channel_desc, n_queryPoints, n_dimentions);

    if (error != hipSuccess)
    {

        printf("Error in hipMallocArray: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);

        return false;
    }

    // Copy query points from host to device
    error = hipMemcpyToArray(query_array_dev, 0, 0, query_h, n_queryPoints * sizeof(float) * n_dimentions, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {

        printf("Error in hipMemcpyToArray: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);

        return false;
    }

    // Resource descriptor
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = query_array_dev;

    // Texture descriptor
    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0] = hipAddressModeClamp;
    tex_desc.addressMode[1] = hipAddressModeClamp;
    tex_desc.filterMode = hipFilterModePoint;
    tex_desc.readMode = hipReadModeElementType;
    tex_desc.normalizedCoords = 0;

    hipTextureObject_t query_tex_dev = 0;
    error = hipCreateTextureObject(&query_tex_dev, &res_desc, &tex_desc, NULL);

    if (error != hipSuccess)
    {

        printf("Error in hipCreateTextureObject: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);

        return false;
    }

    /////only considered >16
    int block_size_x = warpSize / 2;
    int block_size_y = warpSize / 2;
    int grid_size_x = n_refPoints / block_size_x;
    int grid_size_y = n_queryPoints / block_size_y;

    dim3 block_size = dim3(block_size_x, block_size_y);
    dim3 grid_size = dim3(grid_size_x, grid_size_y);

    calc_dist_texture<<<grid_size, block_size>>>(query_tex_dev, n_queryPoints, ref_dev, n_refPoints_original, ref_pitch, n_dimentions, dist_dev, dist_pitch);

    // hipDeviceSynchronize();
    hipDeviceSynchronize();

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        printf("Error in calc_dist_texture kernel: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);
        hipDestroyTextureObject(query_tex_dev);

        return false;
    }

    //////////////////////////////////////////////////////////////////////////////////

    // printf("\n(texture new)before hipMemcpy2D \n");

    //remove after test.....
    
    // error = hipMemcpy2D(dist_h, n_refPoints * sizeof(float), dist_dev, dist_pitch_in_bytes, n_refPoints * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);
    // error = hipMemcpy2D(dist_h, n_refPoints_original * sizeof(float), dist_dev, dist_pitch_in_bytes, n_refPoints_original * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);

    // printf("\n(texture new)after hipMemcpy2D \n");

    // if (error != hipSuccess)
    // {
    //     // printf("error in calc_dist_texture\n");
    //     printf("Error hipMemcpy2D hipMemcpyDeviceToHost after calc_dist_texture kernel execution: %s\n", hipGetErrorString(error));
    //     // exit(-1);
    //     hipFree(ref_dev);
    //     hipFree(dist_dev);
    //     hipFree(idx_dev);
    //     hipFreeArray(query_array_dev);
    //     hipDestroyTextureObject(query_tex_dev);

    //     return false;
    // }


    ////////////////////////////////////////////////////////////////////////////////////////

    block_size_x = (n_refPoints / 2) / warpSize;
    block_size_y = n_queryPoints / warpSize;

    block_size = dim3(warpSize, warpSize);
    grid_size = dim3(block_size_x, block_size_y);

    //////////////////////////////////////////////////

     unsigned int numStages = 0, stage = 0, passOfStage = 0, temp = 0;

    for (temp = n_refPoints; temp > 1; temp >>= 1)
    {
        ++numStages;
    }

    for (stage = 0; stage < numStages; ++stage)
    {

        for (passOfStage = 0; passOfStage < stage + 1; ++passOfStage)
        {
            sort_dist_bitonic<<<grid_size, block_size>>>(dist_dev, idx_dev, n_refPoints, dist_pitch, n_queryPoints, stage, passOfStage);
            hipDeviceSynchronize();
        }
    }

    error = hipGetLastError();

    if (error != hipSuccess)

    {
        printf("Error in sort_dist_bitonic kernel: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);
        hipDestroyTextureObject(query_tex_dev);

        return false;
    }

    error = hipMemcpy2D(dist_h, k * sizeof(float), dist_dev, dist_pitch_in_bytes, k * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);
    error = hipMemcpy2D(idx_h, k * sizeof(int), idx_dev, idx_pitch_in_bytes, k * sizeof(int), n_queryPoints, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("Error in hipMemcpy2D hipMemcpyDeviceToHost after sort_dist_bitonic kernel execution: %s\n", hipGetErrorString(error));
        hipFree(ref_dev);
        hipFree(dist_dev);
        hipFree(idx_dev);
        hipFreeArray(query_array_dev);
        hipDestroyTextureObject(query_tex_dev);

        return false;
    }

     // Memory clean-up
     hipFree(ref_dev);
     hipFree(dist_dev);
     hipFree(idx_dev);
     hipFreeArray(query_array_dev);
     hipDestroyTextureObject(query_tex_dev);
 
     return true;

}
