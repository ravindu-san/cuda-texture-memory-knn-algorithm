#include "hip/hip_runtime.h"
// #include <stdlib.h>
#include <stdio.h>
#include "utilities.h"

//ToDo
//kernel execution
//copy distance values and check

__global__ void calc_dist_texture(hipTextureObject_t queryP,
                                  int n_queryP,
                                  float *refP,
                                  int n_refP,
                                  int ref_pitch,
                                  int n_dim,
                                  float *dist)
{
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex < n_refP && yIndex < n_queryP)
    {
        float ssd = 0.f;
        for (int i = 0; i < n_dim; i++)
        {
            // float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            float tmp = refP[i * ref_pitch + xIndex] - tex2D<float>(queryP, (float)yIndex, (float)i);
            ssd += tmp * tmp;
        }
        // dist[yIndex * query_pitch + xIndex] = ssd;
        dist[yIndex * ref_pitch + xIndex] = ssd;
    }
}

int main()
{

    // int n_refPoints = 8192;
    // int n_queryPoints = 1024;
    int n_refPoints = 16;
    int n_queryPoints = 2;
    int n_dimentions = 4;
    int k = 4;
    int n_clases = 2;
    int clsOfQuerypts[n_queryPoints];

    hipError_t error;
    hipDeviceProp_t prop;
    int device_count;
    int warpSize = 32;

    error = hipGetDeviceCount(&device_count);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    error = hipGetDeviceProperties(&prop, 0);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    warpSize = prop.warpSize;


    float *ref_row_maj_h = (float *)malloc(sizeof(float) * n_dimentions * n_refPoints);
    float *ref_h = (float *)malloc(sizeof(float) * n_dimentions * n_refPoints);
    float *dist_h = (float *)malloc(sizeof(float) * n_refPoints * n_queryPoints);
    int *cls_h = (int *)malloc(sizeof(int) * n_refPoints * n_queryPoints);
    float *query_row_maj_h = (float *)malloc(sizeof(float) * n_dimentions * n_queryPoints);
    float *query_h = (float *)malloc(sizeof(float) * n_dimentions * n_queryPoints);

    char *refPointsFileName = "testData32_4.csv";
    char *queryPointsFileName = "queryPoints1_4.csv";

    readRefPoints(refPointsFileName, ref_row_maj_h, cls_h, n_refPoints, n_queryPoints, n_dimentions);

    ref_h = transpose(ref_row_maj_h, n_refPoints, n_dimentions); //make column major
    free(ref_row_maj_h);

    // for (int i = 0; i < noOfRefPoints; i++)
    for (int i = 0; i < 5; i++)
    {
        printf("%d  %f  %f  %f  %f  %d\n", i, ref_h[i * n_dimentions + 0], ref_h[i * n_dimentions + 1], ref_h[i * n_dimentions + 2], ref_h[i * n_dimentions + 3], cls_h[i]);
    }

    readQueryPoints(queryPointsFileName, query_row_maj_h, n_dimentions);
    query_h = transpose(query_row_maj_h, n_queryPoints, n_dimentions); //make column major
    free(query_row_maj_h);

    

    // Allocate global memory
    float *ref_dev = NULL;
    float *dist_dev = NULL;
    int *cls_dev = NULL;

    size_t ref_pitch_in_bytes;
    size_t dist_pitch_in_bytes;
    size_t cls_pitch_in_bytes;

    error = hipMallocPitch((void **)&ref_dev, &ref_pitch_in_bytes, n_refPoints * sizeof(float), n_dimentions);
    error = hipMallocPitch((void **)&dist_dev, &dist_pitch_in_bytes, n_refPoints * sizeof(float), n_queryPoints);
    error = hipMallocPitch((void **)&cls_dev, &cls_pitch_in_bytes, n_refPoints * sizeof(int), n_queryPoints);

    if (error != hipSuccess)
    {
        printf("Error in hipMallocPitch: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Deduce pitch value of reference points
    size_t ref_pitch = ref_pitch_in_bytes / sizeof(float);
    size_t dist_pitch = dist_pitch_in_bytes / sizeof(float);
    size_t cls_pitch = cls_pitch_in_bytes / sizeof(int);

    //copy ref data from host to device (in column major)
    error = hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_h, n_refPoints * sizeof(float), n_refPoints * sizeof(float), n_dimentions, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {

        printf("Error in hipMemcpy2D: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Allocate CUDA array for query points
    hipArray *query_array_dev = NULL;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    error = hipMallocArray(&query_array_dev, &channel_desc, n_queryPoints, n_dimentions);

    if (error != hipSuccess)
    {

        printf("Error in hipMemcpy2D: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Copy query points from host to device
    error = hipMemcpyToArray(query_array_dev, 0, 0, query_h, n_queryPoints * sizeof(float) * n_dimentions, hipMemcpyHostToDevice);

    // Resource descriptor
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = query_array_dev;

    // Texture descriptor
    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0] = hipAddressModeClamp;
    tex_desc.addressMode[1] = hipAddressModeClamp;
    tex_desc.filterMode = hipFilterModePoint;
    tex_desc.readMode = hipReadModeElementType;
    tex_desc.normalizedCoords = 0;

    hipTextureObject_t query_tex_dev = 0;
    error = hipCreateTextureObject(&query_tex_dev, &res_desc, &tex_desc, NULL);

    printf("\ntexture object created...\n");

    
    int block_size_x = warpSize/2;
    int block_size_y = warpSize/2;
    int grid_size_x = n_refPoints /block_size_x;
    int grid_size_y = n_queryPoints/block_size_y;

    // dim3 block_size = dim3(block_size_x, block_size_y);
    // dim3 grid_size = dim3(grid_size_x, grid_size_y);

    dim3 block_size = dim3(16, 2);
    dim3 grid_size = dim3(n_refPoints/16, 1);


    // calc_dist_texture<<<grid_size, block_size>>>(query_tex_dev, n_queryPoints, ref_dev, n_refPoints, ref_pitch, n_dimentions, dist_dev);
    calc_dist_texture<<<grid_size, block_size>>>(query_tex_dev, n_queryPoints, ref_dev, n_refPoints, ref_pitch, n_dimentions, dist_dev);

    hipDeviceSynchronize();
    // hipDeviceSynchronize();

    error = hipGetLastError();

    if (error != hipSuccess)

    {
        printf("error in kernel\n");
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    error = hipMemcpy2D(dist_h,  n_refPoints * sizeof(float), dist_dev,  dist_pitch_in_bytes,  n_refPoints * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);

    for(int i=0; i< n_refPoints;i++){
        printf("%f  ", dist_h[i]);
    }

    hipFree(ref_dev);
    hipFree(dist_dev);
    hipFree(cls_dev);
    hipFreeArray(query_array_dev);
    free(ref_h);
    free(dist_h);
    free(cls_h);

    return 0;
}