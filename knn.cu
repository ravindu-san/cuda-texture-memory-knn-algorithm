#include "hip/hip_runtime.h"
//
// Created by ravindu on 2020-11-15.
//

//#include <stdlib.h>
#include <stdio.h>
#include "utilities.h"

__global__ void calc_dist_global_mem(float *refP, float *queryP, float *distances, int n_refP, int n_queryP, int n_dim)
{

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //ref points vary across x axis of grid
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //query points vary across y axis of grid

    float sqrd_dist;

    for (int i = 0; i < n_dim; i++)
    {

        float diff = refP[xIndex * n_dim + i] - queryP[yIndex * n_dim + i];//ref points & query points are in row major order
        sqrd_dist += diff * diff;
    }

    distances[yIndex * n_refP + xIndex] = sqrd_dist;

    // if(xIndex == 0 && yIndex == 0){
        printf("distance of ref %d q %d : %f\n", xIndex, yIndex,sqrd_dist);
    // }
}

int main()
{

    int n_refPoints = 8192;
    int n_queryPoints = 2048;
    int n_dimentions = 4;

    float *refPoints_h, *refPoints_d;
    // ClassAndDist *classAndDistArr_h, *classAndDistArr_d;
    int *clases_h, *clases_d;
    float *queryPoints_h, *queryPoints_d;

    float *distances_d;

    hipError_t error;
    hipDeviceProp_t prop;
    int device_count;
    int warpSize = 32;

    error = hipGetDeviceCount(&device_count);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    error = hipGetDeviceProperties(&prop, 0);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    warpSize = prop.warpSize;

    printf("device count : %d\n", device_count);
    printf("device name : %s\n", prop.name);
    printf("device total global memory(KB) : %d\n", prop.totalGlobalMem / 1024);
    printf("max texture dimension x : %d    y : %d\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);

    refPoints_h = (float *)malloc(sizeof(float) * n_dimentions * n_refPoints);
    // classAndDistArr_h = (ClassAndDist *)malloc(sizeof(ClassAndDist) * n_refPoints * n_queryPoints);
    clases_h = (int *) malloc(sizeof(int) * n_refPoints * n_queryPoints);
    queryPoints_h = (float *)malloc(sizeof(float) * n_dimentions * n_queryPoints);

    char *refPointsFileName = "testData8192_4.csv";
    char *queryPointsFileName = "queryPoints_4.csv";

    readRefPoints(refPointsFileName, refPoints_h, clases_h, n_refPoints, n_queryPoints, n_dimentions);

    // for (int i = 0; i < noOfRefPoints; i++)
    // for (int i = 0; i < 5; i++)
    // {
    //     printf("%d  %f  %f  %f  %f  %d\n", i, refPoints_h[i*n_dimentions + 0], refPoints_h[i*n_dimentions + 1], refPoints_h[i*n_dimentions + 2], refPoints_h[i*n_dimentions + 3], classAndDistArr_h[i].cls);
    // }

    readQueryPoints(queryPointsFileName, queryPoints_h, n_dimentions);

    // for (int i = 0; i < n_queryPoints; i++)
    // {
    //     printf("%d  %f  %f  %f  %f \n", i, queryPoints_h[i*n_dimentions + 0], queryPoints_h[i * n_dimentions + 1], queryPoints_h[i*n_dimentions + 2], queryPoints_h[i*n_dimentions + 3]);
    // }


    error = hipMalloc((void **)&refPoints_d, sizeof(float) * n_dimentions * n_refPoints);
    error = hipMalloc((void **)&queryPoints_d, sizeof(float) * n_dimentions * n_queryPoints);
    // // error = hipMalloc((void **)classAndDistArr_d, sizeof(ClassAndDist) * n_refPoints * n_queryPoints);
    error = hipMalloc((void **)&clases_d, sizeof(int) * n_refPoints * n_queryPoints);
    error = hipMalloc((void **)&distances_d, sizeof(float) * n_refPoints * n_queryPoints);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMemcpy(refPoints_d, refPoints_h, sizeof(float) * n_dimentions * n_refPoints, hipMemcpyHostToDevice);
    hipMemcpy(queryPoints_d, queryPoints_h, sizeof(float) * n_dimentions * n_queryPoints, hipMemcpyHostToDevice);
    hipMemcpy(clases_d, clases_h, sizeof(int) * n_refPoints * n_queryPoints, hipMemcpyHostToDevice);


    int block_dim = warpSize / 2;
    int grid_dim = (n_refPoints / block_dim);

    dim3 block_size = dim3(block_dim, block_dim);
    dim3 grid_size = dim3(grid_dim, grid_dim);

    printf("\nhello before\n");

    calc_dist_global_mem<<<grid_size, block_size>>>(refPoints_d, queryPoints_d, distances_d, n_refPoints, n_queryPoints, n_dimentions);

    hipDeviceSynchronize();
    // hipDeviceSynchronize();
    
    printf("after kernel");
    // hipMemcpy(classAndDistArr_h, classAndDistArr_d, sizeof(ClassAndDist) * n_refPoints * n_queryPoints, hipMemcpyDeviceToHost);

    // for(int i = 0; i<1 ; i++){

    //     printf("hello");

    //     printf("\n%f\n",_h[0].distance);

    // }

    hipFree(refPoints_d);
    hipFree(queryPoints_d);
    free(refPoints_h);
    free(queryPoints_h);
    free(clases_h);

    return 0;
}
