//
// Created by ravindu on 2020-11-15.
//

//#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){

    hipError_t error;
    hipDeviceProp_t prop;
    int device_count;

    error = hipGetDeviceCount (&device_count);

    if(error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    error = hipGetDeviceProperties(&prop, 0);

    printf("device count : %d\n", device_count);
    printf("device name : %s\n", prop.name);
    printf("device total global memory(KB) : %d\n", prop.totalGlobalMem/1024);
    printf("max texture dimension x : %d    y : %d\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);

    return 0;
}

