#include "hip/hip_runtime.h"
//#include <stdlib.h>
#include <stdio.h>
#include "utilities.h"
// #include <cmath>
// const float infinity = INFINITY;

__global__ void calc_dist_global_mem(float *refP, float *queryP, float *distances, int n_refP_original, int n_refP, int n_queryP, int n_dim)
{

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //ref points vary across x axis of grid
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //query points vary across y axis of grid

    if(xIndex < n_refP_original && yIndex < n_queryP){

        float sqrd_dist;

        for (int i = 0; i < n_dim; i++)
        {

            float diff = refP[xIndex * n_dim + i] - queryP[yIndex * n_dim + i];
            sqrd_dist += diff * diff;
        }

        distances[yIndex * n_refP + xIndex] = sqrd_dist;

    }else if(yIndex < n_queryP)
    {
        distances[yIndex * n_refP + xIndex] = infinity;
    }
    
}



__global__ void sort_dist_bitonic(float *distances, int *indexes, int n_refP, int n_queryP,const uint stage, const uint passOfStage){

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(xIndex < n_refP/2 && yIndex < n_queryP){

        unsigned int pairDistance = 1 << (stage - passOfStage);
        unsigned int blockWidth = 2 * pairDistance;
        unsigned int temp;
        bool compareResult;
    
        unsigned int leftId = (xIndex & (pairDistance - 1)) + (xIndex >> (stage - passOfStage)) * blockWidth;
        unsigned int rightId = leftId + pairDistance;
    
        float leftElement, rightElement;
        float greater, lesser;
        int left_idx, right_idx, greater_idx, lesser_idx;

        leftElement = distances[yIndex * n_refP + leftId];
        rightElement =distances[yIndex * n_refP +rightId];
    
        if (stage == 0 && passOfStage == 0)
        {
            left_idx = leftId;
            right_idx = rightId;
        }
        else
        {

            left_idx = indexes[yIndex * n_refP + leftId];
            right_idx = indexes[yIndex * n_refP + rightId];
        }

        unsigned int sameDirectionBlockWidth = xIndex >> stage;
        unsigned int sameDirection = sameDirectionBlockWidth & 0x1;
    
        temp = sameDirection ? rightId : temp;
        rightId = sameDirection ? leftId : rightId;
        leftId = sameDirection ? temp : leftId;
    
        compareResult = (leftElement < rightElement);
    
        greater = compareResult ? rightElement : leftElement;
        greater_idx = compareResult ? right_idx : left_idx;

        lesser = compareResult ? leftElement : rightElement;
        lesser_idx = compareResult ? left_idx : right_idx;

        distances[yIndex * n_refP + leftId] = lesser;
        distances[yIndex * n_refP +rightId] = greater;
    
        indexes[yIndex * n_refP + leftId] = lesser_idx;
        indexes[yIndex * n_refP + rightId] = greater_idx;

    }
   

}


bool knn_cuda_global(const float *ref_h,
                    int n_refPoints_original,
                    const float *query_h,
                    int n_queryPoints,
                    int n_dimentions,
                    int k,
                    float *dist_h,
                    int *idx_h){

    
    hipError_t error;
    hipDeviceProp_t prop;
    int n_devices;
    int warpSize = 32;

    unsigned int n_refPoints = getNearestIntOfPow2(n_refPoints_original);

    error = hipGetDeviceCount(&n_devices);
    if (error != hipSuccess || n_devices == 0)
    {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    error = hipSetDevice(0);
    if (error != hipSuccess)
    {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    error = hipGetDeviceProperties(&prop, 0);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        return false;
    }

    warpSize = prop.warpSize;

    float *refPoints_d;
    int *idx_dev;
    float *queryPoints_d;
    float *distances_d;

    error = hipMalloc((void **)&refPoints_d, sizeof(float) * n_dimentions * n_refPoints_original);
    error = hipMalloc((void **)&queryPoints_d, sizeof(float) * n_dimentions * n_queryPoints);
    error = hipMalloc((void **)&idx_dev, sizeof(int) * n_refPoints * n_queryPoints);
    error = hipMalloc((void **)&distances_d, sizeof(float) * n_refPoints * n_queryPoints);


    if (error != hipSuccess)
    {
        printf("(global) Error in hipMalloc: %s\n", hipGetErrorString(error));
        hipFree(refPoints_d);
        hipFree(queryPoints_d);
        hipFree(distances_d);
        hipFree(idx_dev);
    }

    error = hipMemcpy(refPoints_d, ref_h, sizeof(float) * n_dimentions * n_refPoints_original, hipMemcpyHostToDevice);
    error = hipMemcpy(queryPoints_d, query_h, sizeof(float) * n_dimentions * n_queryPoints, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("(global) Error in hipMemcpy: %s\n", hipGetErrorString(error));
        hipFree(refPoints_d);
        hipFree(queryPoints_d);
        hipFree(distances_d);
        hipFree(idx_dev);
    }

    /////only considered >16
    int block_size_x = warpSize / 2;
    int block_size_y = warpSize / 2;
    int grid_size_x = n_refPoints / block_size_x;
    int grid_size_y = n_queryPoints / block_size_y;

    dim3 block_size = dim3(block_size_x, block_size_y);
    dim3 grid_size = dim3(grid_size_x, grid_size_y);


    calc_dist_global_mem<<<grid_size, block_size>>>(refPoints_d, queryPoints_d, distances_d, n_refPoints_original, n_refPoints, n_queryPoints, n_dimentions);

    hipDeviceSynchronize();
    // hipDeviceSynchronize();

    error = hipGetLastError();

    if (error != hipSuccess)

    {
        printf("(global) Error in calc_dist_global_mem: %s\n", hipGetErrorString(error));
        hipFree(refPoints_d);
        hipFree(queryPoints_d);
        hipFree(distances_d);
        hipFree(idx_dev);
    }


    grid_size_x = (n_refPoints / 2) / warpSize;
    grid_size_y = n_queryPoints / warpSize;

    block_size = dim3(warpSize, warpSize);
    grid_size = dim3(grid_size_x, grid_size_y);


    unsigned int numStages = 0, stage = 0, passOfStage = 0, temp = 0;

    for (temp = n_refPoints; temp > 1; temp >>= 1)
    {
        ++numStages;
    }

    for (stage = 0; stage < numStages; ++stage)
    {

        for (passOfStage = 0; passOfStage < stage + 1; ++passOfStage)
        {

            sort_dist_bitonic<<<grid_size, block_size>>>(distances_d, idx_dev, n_refPoints, n_queryPoints, stage, passOfStage);
            hipDeviceSynchronize();
        }
    }

    error = hipGetLastError();

    if (error != hipSuccess)

    {
         printf("(global) Error in sort_dist_bitonic kernel: %s\n", hipGetErrorString(error));
         hipFree(refPoints_d);
         hipFree(queryPoints_d);
         hipFree(distances_d);
         hipFree(idx_dev);
 
         return false;
    }

    error = hipMemcpy2D(dist_h, k * sizeof(float), distances_d, n_refPoints*sizeof(float), k * sizeof(float), n_queryPoints, hipMemcpyDeviceToHost);
    error = hipMemcpy2D(idx_h, k * sizeof(int), idx_dev, n_refPoints*sizeof(int), k * sizeof(int), n_queryPoints, hipMemcpyDeviceToHost);


    if (error != hipSuccess)

    {
         printf("(global) Error in hipMemcpy or hipMemcpy2D: %s\n", hipGetErrorString(error));
         hipFree(refPoints_d);
         hipFree(queryPoints_d);
         hipFree(distances_d);
         hipFree(idx_dev);
 
         return false;
    }
    
    hipFree(refPoints_d);
    hipFree(queryPoints_d);
    hipFree(distances_d);
    hipFree(idx_dev);

    return true;


}
