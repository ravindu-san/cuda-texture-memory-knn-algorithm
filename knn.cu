#include "hip/hip_runtime.h"
//
// Created by ravindu on 2020-11-15.
//

//#include <stdlib.h>
#include <stdio.h>
#include "utilities.h"

__global__ void calc_dist_global_mem(float *refP, float *queryP, float *distances, int n_refP, int n_queryP, int n_dim)
{

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x; //ref points vary across x axis of grid
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y; //query points vary across y axis of grid

    if(xIndex < n_refP && yIndex < n_queryP){

        float sqrd_dist;

        for (int i = 0; i < n_dim; i++)
        {

            float diff = refP[xIndex * n_dim + i] - queryP[yIndex * n_dim + i];//ref points & query points are in row major order
            sqrd_dist += diff * diff;
        }

        distances[yIndex * n_refP + xIndex] = sqrd_dist;

        if(xIndex == 0 && yIndex == 1023){
            printf("distance of ref %d q %d : %f\n", xIndex, yIndex,sqrd_dist);
        }

    }
    
}

__global__ void sort_dist_bitonic(float *distances, int *clases, int n_refP, int n_queryP,const uint stage, const uint passOfStage){

    // uint threadId = get_global_id(0);
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(xIndex < n_refP/2 && yIndex < n_queryP){

        unsigned int pairDistance = 1 << (stage - passOfStage);
        unsigned int blockWidth = 2 * pairDistance;
        unsigned int temp;
        bool compareResult;
    
        unsigned int leftId = (xIndex & (pairDistance - 1)) + (xIndex >> (stage - passOfStage)) * blockWidth;
        unsigned int rightId = leftId + pairDistance;
    
        float leftElement, rightElement;
        float greater, lesser;
        int leftElement_cls, rightElement_cls, greater_cls, lesser_cls;

        leftElement = distances[yIndex * n_refP + leftId];
        rightElement =distances[yIndex * n_refP +rightId];
    
        leftElement_cls = clases[yIndex * n_refP + leftId];
        rightElement_cls = clases[yIndex * n_refP +rightId];
    
        unsigned int sameDirectionBlockWidth = xIndex >> stage;
        unsigned int sameDirection = sameDirectionBlockWidth & 0x1;
    
        temp = sameDirection ? rightId : temp;
        rightId = sameDirection ? leftId : rightId;
        leftId = sameDirection ? temp : leftId;
    
        compareResult = (leftElement < rightElement);
    
    /////////////////////////////////////////////////////////////////////////////    
        /*add these to a single if else block*/
        greater = compareResult ? rightElement : leftElement;
        greater_cls = compareResult ? rightElement_cls : leftElement_cls;
        lesser = compareResult ? leftElement : rightElement;
        lesser_cls = compareResult ? leftElement_cls : rightElement_cls;
    //////////////////////////////////////////////////////////////////////////////

        distances[yIndex * n_refP + leftId] = lesser;
        distances[yIndex * n_refP +rightId] = greater;
    
        clases[yIndex * n_refP + leftId] = lesser_cls;
        clases[yIndex * n_refP +rightId] = greater_cls;
    }
   

}



int main()
{

    // int n_refPoints = 8192;
    // int n_queryPoints = 1024;
    int n_refPoints = 16;
    int n_queryPoints = 2;
    int n_dimentions = 4;

    float *refPoints_h, *refPoints_d;
    // ClassAndDist *classAndDistArr_h, *classAndDistArr_d;
    int *clases_h, *clases_d;
    float *queryPoints_h, *queryPoints_d;

    float *distances_h, *distances_d;//distances_h not needed..only for test

    hipError_t error;
    hipDeviceProp_t prop;
    int device_count;
    int warpSize = 32;

    error = hipGetDeviceCount(&device_count);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    error = hipGetDeviceProperties(&prop, 0);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    warpSize = prop.warpSize;

    printf("device count : %d\n", device_count);
    printf("device name : %s\n", prop.name);
    printf("device total global memory(KB) : %d\n", prop.totalGlobalMem / 1024);
    printf("max texture dimension x : %d    y : %d\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);

    refPoints_h = (float *)malloc(sizeof(float) * n_dimentions * n_refPoints);
    // classAndDistArr_h = (ClassAndDist *)malloc(sizeof(ClassAndDist) * n_refPoints * n_queryPoints);
    clases_h = (int *) malloc(sizeof(int) * n_refPoints * n_queryPoints);
    queryPoints_h = (float *)malloc(sizeof(float) * n_dimentions * n_queryPoints);

    distances_h = (float *)malloc(sizeof(float)*n_refPoints*n_queryPoints);

    // char *refPointsFileName = "testData8192_4.csv";
    // char *queryPointsFileName = "queryPoints_4.csv";
     char *refPointsFileName = "testData16_4.csv";
    char *queryPointsFileName = "queryPoints1_4.csv";

    readRefPoints(refPointsFileName, refPoints_h, clases_h, n_refPoints, n_queryPoints, n_dimentions);

    // for (int i = 0; i < noOfRefPoints; i++)
    // for (int i = 0; i < 5; i++)
    // {
    //     printf("%d  %f  %f  %f  %f  %d\n", i, refPoints_h[i*n_dimentions + 0], refPoints_h[i*n_dimentions + 1], refPoints_h[i*n_dimentions + 2], refPoints_h[i*n_dimentions + 3], classAndDistArr_h[i].cls);
    // }

    readQueryPoints(queryPointsFileName, queryPoints_h, n_dimentions);

    // for (int i = 0; i < n_queryPoints; i++)
    // {
    //     printf("%d  %f  %f  %f  %f \n", i, queryPoints_h[i*n_dimentions + 0], queryPoints_h[i * n_dimentions + 1], queryPoints_h[i*n_dimentions + 2], queryPoints_h[i*n_dimentions + 3]);
    // }


    error = hipMalloc((void **)&refPoints_d, sizeof(float) * n_dimentions * n_refPoints);
    error = hipMalloc((void **)&queryPoints_d, sizeof(float) * n_dimentions * n_queryPoints);
    // // error = hipMalloc((void **)classAndDistArr_d, sizeof(ClassAndDist) * n_refPoints * n_queryPoints);
    error = hipMalloc((void **)&clases_d, sizeof(int) * n_refPoints * n_queryPoints);
    error = hipMalloc((void **)&distances_d, sizeof(float) * n_refPoints * n_queryPoints);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMemcpy(refPoints_d, refPoints_h, sizeof(float) * n_dimentions * n_refPoints, hipMemcpyHostToDevice);
    hipMemcpy(queryPoints_d, queryPoints_h, sizeof(float) * n_dimentions * n_queryPoints, hipMemcpyHostToDevice);
    hipMemcpy(clases_d, clases_h, sizeof(int) * n_refPoints * n_queryPoints, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    int block_dim = warpSize / 2;
    int grid_dim = (n_refPoints / block_dim);

    dim3 block_size = dim3(block_dim, block_dim);
    dim3 grid_size = dim3(grid_dim, grid_dim);

    printf("\nhello before\n");

    calc_dist_global_mem<<<grid_size, block_size>>>(refPoints_d, queryPoints_d, distances_d, n_refPoints, n_queryPoints, n_dimentions);

    hipDeviceSynchronize();
    // hipDeviceSynchronize();

    error = hipGetLastError();

    if (error != hipSuccess)

    {
        printf("error in kernel\n");
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }


    
    printf("after kernel");
    error = hipMemcpy(distances_h, distances_d, sizeof(float) * n_refPoints * n_queryPoints, hipMemcpyDeviceToHost);


    if (error != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }


    printf("distances before sort\n");
    for(int i = 0; i<n_refPoints ; i++){

        // printf("hello");

        printf("%f  ", i,distances_h[n_refPoints + i]);

    }


    /////////////////////////////////////////////////////////////////////////////////

    // int block_count_x = (n_refPoints / 2) / warpSize;
    // int block_count_y = n_queryPoints / warpSize;
    

    // block_size = dim3(warpSize, warpSize);
    // block_size = dim3(warpSize, warpSize);
    // grid_size = dim3(block_count_x, block_count_y);

    block_size = dim3(8, 2);
    grid_size = dim3(1, 1);


    unsigned int numStages = 0, stage = 0, passOfStage = 0, temp = 0;

    for (temp = n_refPoints; temp > 1; temp >>= 1)
    {
        ++numStages;
    }

    for (stage = 0; stage < numStages; ++stage)
    {

        for (passOfStage = 0; passOfStage < stage + 1; ++passOfStage)
        {

            sort_dist_bitonic<<<grid_size, block_size>>>(distances_d, clases_d, n_refPoints, n_queryPoints, stage, passOfStage);
            hipDeviceSynchronize();
        }
    }

    error = hipGetLastError();

    if (error != hipSuccess)

    {
        printf("error in sort kernel\n");
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    error = hipMemcpy(distances_h, distances_d, sizeof(float) * n_refPoints * n_queryPoints, hipMemcpyDeviceToHost);
    
    printf("\n\ndistances after sort\n");
    for(int i = 0; i<n_refPoints ; i++){

        printf("%f  ", i,distances_h[n_refPoints + i]);

    }
    /////////////////////////////////////////////////////////////////////////////////

    hipFree(refPoints_d);
    hipFree(queryPoints_d);
    hipFree(clases_d);
    hipFree(distances_d);
    free(refPoints_h);
    free(queryPoints_h);
    free(clases_h);
    free(distances_h);//not need if distances are not get back to host

    return 0;
}
